#include <stdio.h>
#include <hip/hip_runtime.h>
__device__ char xx;

__global__ void cuCopyTest( char *s1, char *s2)
{
    char out;
    char * dest;
    char * src;
    int n;

    dest = &xx;
    src = s1;
    n = 1;
    while(n-- > 0)
        *dest++ = *src++;

    dest = &out;
    src = s2;
    n = 1;
    while(n-- > 0)
        *dest++ = *src++;

}


