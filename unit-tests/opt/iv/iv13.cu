
#include <hip/hip_runtime.h>
/*
 * a simple test
 */

__shared__ float data1[32][32];
__shared__ float data2[32][32];
__shared__ float data3[32][32];

__device__ void mult(__shared__ float d1[32][32],
                     __shared__ float d2[32][32],
                     __shared__ float d3[32][32],
                     int idx) 
{
  int i;
  int j, k, l;
  j = -1;
  k = 0;
  l = 1;
  for (i = 0; i < 31; i+=2) {

    d1[idx][i] = d2[idx+1][j] + d2[idx][j] + d2[idx-1][j] +
                 d2[idx+1][k] + d2[idx][k] + d2[idx-1][k] + 
                 d2[idx+1][l] + d2[idx][l] + d2[idx-1][l];

    d1[idx][i+1] = d2[idx+1][j+1] + d2[idx][j+1] + d2[idx-1][j+1] +
                 d2[idx+1][k+1] + d2[idx][k+1] + d2[idx-1][k+1] + 
                 d2[idx+1][l+1] + d2[idx][l+1] + d2[idx-1][l+1];
    j+=2;
    k+=2;
    l+=2;
  }
}

__global__ void doit(int start, int end) {
  int i;
  for (i = start; i < end; i++) {
    mult(data1, data2, data3, i);
  }
}



